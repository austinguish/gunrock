#include <set>

#include <gunrock/algorithms/color.hxx>
#include "color_cpu.hxx"  // Reference implementation
#include <iostream>
#include <nvToolsExt.h>
#include <cudaProfiler.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
using namespace gunrock;
using namespace memory;

void test_color(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;
  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  std::string filename = argument_array[1];

  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load(filename);

  csr_t csr;

  csr.from_coo(coo);

  // --
  // Build graph

  auto G = graph::build<memory_space_t::device>(properties, csr);

  // --
  // Params and memory allocation

  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<vertex_t> colors(n_vertices);

  // --
  // GPU Run
  hipProfilerStart();
  float gpu_elapsed = gunrock::color::run(G, colors.data().get());
  hipProfilerStop();
  // // --
  // // CPU Run

  // thrust::host_vector<vertex_t> h_colors(n_vertices);

  // float cpu_elapsed =
  //     color_cpu::run<csr_t, vertex_t, edge_t, weight_t>(csr,
  //     h_colors.data());

  // int n_errors = color_cpu::compute_error<csr_t, vertex_t, edge_t, weight_t>(
  //     csr, colors, h_colors);

  // std::vector<int> stl_colors(n_vertices);
  // thrust::copy(colors.begin(), colors.end(), stl_colors.begin());
  // int n_colors = std::set(stl_colors.begin(), stl_colors.end()).size();

  // --
  // Log
  print::head(colors, 40, "GPU colors");
  // print::head(h_colors, 40, "CPU colors");

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
}

int main(int argc, char** argv) {
  test_color(argc, argv);
}
